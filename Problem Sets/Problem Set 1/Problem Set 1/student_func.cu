#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

const size_t block_x = 32;
const size_t block_y = 32;

	__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
		unsigned char* const greyImage,
		int numRows, int numCols)
{
	//TODO
	//Fill in the kernel to convert from color to greyscale
	//the mapping from components of a uchar4 to RGBA is:
	// .x -> R ; .y -> G ; .z -> B ; .w -> A
	//
	//The output (greyImage) at each pixel should be the result of
	//applying the formula: output = .299f * R + .587f * G + .114f * B;
	//Note: We will be ignoring the alpha channel for this conversion

	//First create a mapping from the 2D block and grid locations
	//to an absolute 2D location in the image, then use that to
	//calculate a 1D offset
	size_t blkid_x = blockIdx.x;
	size_t blkid_y = blockIdx.y;
	size_t thid_x = threadIdx.x;
	size_t thid_y = threadIdx.y;
	int tpbx = numCols/block_x + 1;
	//int tpby = numRows/block_y;
	size_t realid = (blkid_x + blkid_y*tpbx)*block_x*block_y + thid_x + thid_y*block_x;
	if (realid < numRows*numCols){
		uchar4 rgba = rgbaImage[realid];
		float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
		greyImage[realid] = channelSum;
	}
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
		unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
	//You must fill in the correct sizes for the blockSize and gridSize
	//currently only one block with one thread is being launched
	const dim3 blockSize(block_x, block_y, 1);  //TODO
	int gridx = numCols/block_x + 1;
	int gridy = numRows/block_y + 1;
	printf("grid x is %d\ngrid y is %d\n", gridx, gridy);
	const dim3 gridSize(gridx, gridy, 1);  //TODO

	rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
